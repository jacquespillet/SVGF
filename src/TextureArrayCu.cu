#include "hip/hip_runtime.h"
#include "TextureArrayCu.cuh"
#include <iostream>
#include <assert.h>

#define CUDA_CHECK_ERROR(err) \
    do { \
        hipError_t error = err; \
        if (error != hipSuccess) { \
            std::cout << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(error) << std::endl; \
            assert(false); \
        } \
    } while (0)
namespace gpupt
{
    textureArrayCu::~textureArrayCu() {
        hipDeviceSynchronize();
        
        hipDestroyTextureObject(TexObject);
        hipFree(CuArray);   
        
    }


    void textureArrayCu::CreateTextureArray(int Width, int Height, int Layers, bool IsFloat) {
        this->Width = Width;
        this->Height = Height;

        if(IsFloat)
        {
            hipMallocPitch((void**)&CuArray, &Pitch,  TotalWidth*sizeof(float4), TotalHeight);
            struct hipResourceDesc resDesc;
            memset(&resDesc, 0, sizeof(resDesc));
            resDesc.resType = hipResourceTypePitch2D;
            resDesc.res.pitch2D.devPtr = CuArray;
            resDesc.res.pitch2D.width = TotalWidth;
            resDesc.res.pitch2D.height = TotalHeight;
            resDesc.res.pitch2D.desc = hipCreateChannelDesc<float4>();
            resDesc.res.pitch2D.pitchInBytes = Pitch;
            struct hipTextureDesc texDesc;
            memset(&texDesc, 0, sizeof(texDesc));
            hipCreateTextureObject(&TexObject, &resDesc, &texDesc, NULL);        
        }
        else
        {
            hipMallocPitch((void**)&CuArray, &Pitch,  TotalWidth*sizeof(uchar4), TotalHeight);
            struct hipResourceDesc resDesc;
            memset(&resDesc, 0, sizeof(resDesc));
            resDesc.resType = hipResourceTypePitch2D;
            resDesc.res.pitch2D.devPtr = CuArray;
            resDesc.res.pitch2D.width = TotalWidth;
            resDesc.res.pitch2D.height = TotalHeight;
            resDesc.res.pitch2D.desc = hipCreateChannelDesc<uchar4>();
            resDesc.res.pitch2D.pitchInBytes = Pitch;
            struct hipTextureDesc texDesc;
            texDesc.filterMode = hipTextureFilterMode::hipFilterModeLinear;
            memset(&texDesc, 0, sizeof(texDesc));
            hipCreateTextureObject(&TexObject, &resDesc, &texDesc, NULL);        
        }
    }

    void textureArrayCu::LoadTextureLayer(int layerIndex, const std::vector<uint8_t>& ImageData, int Width, int Height) {
        static int LayersPerRow = TotalWidth / Width;
        int DestInxX = layerIndex % LayersPerRow;
        int DestInxY = layerIndex / LayersPerRow;
        uint32_t DestX = DestInxX * Width;
        uint32_t DestY = DestInxY * Height;
        uint32_t Dest = (DestY * TotalWidth + DestX) * 4;


        hipMemcpy2D((uint8_t*)CuArray + Dest, Pitch, ImageData.data(), Width*sizeof(uchar4), Width*sizeof(uchar4), Height, hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR(hipGetLastError());
    }

    void textureArrayCu::LoadTextureLayer(int layerIndex, const std::vector<float>& ImageData, int Width, int Height) {
        static int LayersPerRow = TotalWidth / Width;
        int DestInxX = layerIndex % LayersPerRow;
        int DestInxY = layerIndex / LayersPerRow;
        uint32_t DestX = DestInxX * Width;
        uint32_t DestY = DestInxY * Height;
        uint32_t Dest = (DestY * TotalWidth + DestX) * 4;


        hipMemcpy2D((float*)CuArray + Dest, Pitch, ImageData.data(), Width*sizeof(float4), Width*sizeof(float4), Height, hipMemcpyHostToDevice);
    }
}